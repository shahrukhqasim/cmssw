#include "hip/hip_runtime.h"
#include "RecoLocalCalo/HGCalRecAlgos/interface/BinnerGPU.h"
#include "HeterogeneousCore/CUDAUtilities/interface/GPUVecArray.h"
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>



namespace BinnerGPU {

typedef GPU::VecArray<int,MAX_DEPTH> RequiredBin;


__global__ void kernel_compute_histogram(RechitForBinning*dInputData, RequiredBin*dOutputData, const size_t numRechits) {

    size_t rechitLocation = blockIdx.x * blockDim.x + threadIdx.x;

    if(rechitLocation >= numRechits)
        return;

    float eta = dInputData[rechitLocation].eta;
    float phi = dInputData[rechitLocation].phi;
    unsigned int index = dInputData[rechitLocation].index;

    int etaIndex = floor((abs(eta) - 1.6) / 0.05);
    int phiIndex = floor((phi + M_PI) / 0.05);

    dOutputData[phiIndex*ETA_BINS + etaIndex].push_back(index);
}



std::shared_ptr<int> computeBins(std::vector<RechitForBinning> layerData) {
    std::shared_ptr<int> hOutputData(new int[ETA_BINS*PHI_BINS*MAX_DEPTH]);

    // Allocate memory and put data into device
    RequiredBin* dOutputData;
    RechitForBinning* dInputData;
    hipMalloc(&dOutputData, sizeof(RequiredBin)*ETA_BINS*PHI_BINS);
    hipMalloc(&dInputData, sizeof(RechitForBinning)*layerData.size());
    hipMemcpy(dInputData, layerData.data(), sizeof(RechitForBinning)*layerData.size(), hipMemcpyHostToDevice);
    hipMemset(dOutputData, 0x00, sizeof(RequiredBin)*ETA_BINS*PHI_BINS);

    // Call the kernel
    const dim3 blockSize(1024,1,1);
    const dim3 gridSize(ceil(layerData.size()/1024.0),1,1);
    kernel_compute_histogram <<<gridSize,blockSize>>>(dInputData, dOutputData, layerData.size());

    // Copy result back!
    hipMemcpy(dOutputData, hOutputData.get(), sizeof(int)*ETA_BINS*PHI_BINS*MAX_DEPTH, hipMemcpyDeviceToHost);

    // Free all the memory
    hipFree(dOutputData);
    hipFree(dInputData);

    
    return hOutputData;
}





__global__ void kernel_search(RechitForBinning*dInputData, RequiredBin*dOutputData, const size_t numRechits, float delta_c) {

    size_t rechitLocation = blockIdx.x * blockDim.x + threadIdx.x;

    size_t rechitLocation1 = rechitLocation / 1024;
    size_t rechitLocation2 = rechitLocation % 1024;

    if(rechitLocation1 >= numRechits or rechitLocation2 >= numRechits)
        return;

    float x1 = dInputData[rechitLocation1].eta;
    float y1 = dInputData[rechitLocation1].phi;

    float x2 = dInputData[rechitLocation2].eta;
    float y2 = dInputData[rechitLocation2].phi;

    if (std::abs(x1-x2) < delta_c && std::abs(y1-y2) < delta_c) {
        dOutputData[rechitLocation1].push_back(dInputData[rechitLocation2].index);
    }
}




std::shared_ptr<int> searchThing(std::vector<RechitForBinning> layerData, float delta_c) {
    std::shared_ptr<int> hOutputData(new int[layerData.size()*MAX_NEIGHBORS]);

    // Allocate memory and put data into device
    RequiredBin* dOutputData;
    RechitForBinning* dInputData;
    hipMalloc(&dOutputData, sizeof(RequiredBin)*MAX_NEIGHBORS);
    hipMalloc(&dInputData, sizeof(RechitForBinning)*layerData.size());
    hipMemcpy(dInputData, layerData.data(), sizeof(RechitForBinning)*layerData.size(), hipMemcpyHostToDevice);
    hipMemset(dOutputData, 0x00, sizeof(RequiredBin)*MAX_NEIGHBORS);

    // Call the kernel
    const dim3 blockSize(1024,1,1);
    const dim3 gridSize(ceil((layerData.size()*layerData.size())/1024.0),1,1);
    kernel_search <<<gridSize,blockSize>>>(dInputData, dOutputData, layerData.size(), delta_c);

    // Copy result back!
    hipMemcpy(dOutputData, hOutputData.get(), sizeof(int)*MAX_NEIGHBORS*MAX_DEPTH, hipMemcpyDeviceToHost);

    // Free all the memory
    hipFree(dOutputData);
    hipFree(dInputData);

    
    return hOutputData;
}

}